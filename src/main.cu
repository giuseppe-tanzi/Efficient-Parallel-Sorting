#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include "../lib/radixSort.cuh"
#include "../lib/mergeSort.cuh"
#include "../lib/parallelSort.cuh"
#include "../lib/utils.cuh"
#include "../lib/utilsParallelSort.cuh"

int main(int argc, char *argv[])
{
    unsigned short n_algorithms = 3;
    char algorithms[][100] = {"Sequential Radix Sort", 
                            "Sequential Merge Sort", 
                            "Parallel Radix Sort + Merge Sort \\w global"};
                            // "Parallel Radix Sort + Merge Sort '\w shared\t(GPU)"};
    char machine[][100] = {"CPU", "CPU", "GPU"};
    unsigned long threads[n_algorithms];
    bool correctness[n_algorithms];
    double elapsed_time[n_algorithms];

    unsigned long long N = 512;
    unsigned short *a, *dev_a;

    // Variables useful for parallel sorting
    ParallelSortConfig sort_config;
    unsigned long lists_to_merge = 0;
    unsigned blocks_involved_in_merging = 0;
    unsigned long long *block_starting_idx;
    unsigned long long *block_size;
    unsigned long *thread_offset;
    unsigned long *dev_thread_offset;

    double t_start = 0, t_stop = 0;

    if (argc > 1)
    {
        N = atoi(argv[1]);
    }

    const size_t size_array = N * sizeof(unsigned short);
    a = (unsigned short *)malloc(size_array);
    cudaHandleError(hipMalloc((void **)&dev_a, size_array));

    // Sequential sorting with Radix Sort
    printf("Sort algorithm on array of %llu elements\n\n", N);
    init_array(a, N);
    t_start = get_time();
    radix_sort(a, N);
    t_stop = get_time();
    threads[0] = 1;
    correctness[0] = is_sorted(a, N);
    elapsed_time[0] = t_stop - t_start;
    bzero(a, size_array); // Erase destination buffer

    // Sequential sorting with Merge Sort
    init_array(a, N);
    t_start = get_time();
    merge_sort(a, 0, N - 1);
    t_stop= get_time();
    threads[1] = 1;
    correctness[1] = is_sorted(a, N);
    elapsed_time[1] = t_stop - t_start;
    bzero(a, size_array); // Erase destination buffer

    // Parallel sorting
    init_array(a, N);
    cudaHandleError(hipMemcpy(dev_a, a, size_array, hipMemcpyHostToDevice));

    t_start = get_time();

    // Determine block and thread configurations
    sort_config = determine_config(N);

    sort_config.blockSize = dim3(sort_config.threads_per_block);
    sort_config.gridSize = dim3(sort_config.total_blocks);

    /*
        - First I get how many sorted list I have at level 0
        - Then I divide by 2 to have the number of merge that I'm going to have at level 0
    */
    lists_to_merge = ceil(get_n_list_to_merge(N, sort_config.partition_size, sort_config.total_threads) / (float)2);

    /*
        - The number of blocks needed during the merging phase
    */
    blocks_involved_in_merging = ceil(lists_to_merge / (float)sort_config.threads_per_block);
    const size_t size_blocks = blocks_involved_in_merging * sort_config.threads_per_block * sizeof(unsigned long);

    /*
        - It contains the start index in the array for each block
    */
    block_starting_idx = (unsigned long long *)malloc(blocks_involved_in_merging * sizeof(unsigned long long));

    /*
        - It contains the size to handle in the data array for each block
    */
    block_size = (unsigned long long *)malloc(blocks_involved_in_merging * sizeof(unsigned long long));

    thread_offset = (unsigned long *)malloc(size_blocks);
    cudaHandleError(hipMalloc((void **)&dev_thread_offset, size_blocks));

    parallel_sort(dev_a, N, sort_config, size_blocks, blocks_involved_in_merging, block_starting_idx, block_size, thread_offset, dev_thread_offset);

    t_stop = get_time();

    cudaHandleError(hipPeekAtLastError());
    cudaHandleError(hipMemcpy(a, dev_a, size_array, hipMemcpyDeviceToHost));

    threads[2] = sort_config.total_threads;
    correctness[2] = is_sorted(a, N);
    elapsed_time[2] = t_stop - t_start;
    bzero(a, size_array); // Erase destination buffer

    // printf("NUM_THREADS: %lu\n", sort_config.total_threads);
    // printf("NUM BLOCKS: %lu\n", sort_config.total_blocks);
    // printf("NUM THREAD PER BLOCK: %lu\n", sort_config.threads_per_block);
    // printf("NUM BLOCKS MERGE: %d\n", blocks_involved_in_merging);
    // printf("PARTITION SIZE: %llu\n", sort_config.partition_size);

    // Print the table
    print_table(n_algorithms, algorithms, machine, threads, correctness, elapsed_time);

    // Cleanup
    free(a);
    free(block_starting_idx);
    free(thread_offset);
    cudaHandleError(hipFree(dev_thread_offset));
    cudaHandleError(hipFree(dev_a));

    return 0;
}