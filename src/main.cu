#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include "../lib/radixSort.cuh"
#include "../lib/mergeSort.cuh"

#define MAXTHREADSPERBLOCK 512
#define MAXBLOCKS 65535

/*
    Useful to check errors in the cuda kernels
*/
#define cudaHandleError(ans)                  \
    {                                         \
        gpuAssert((ans), __FILE__, __LINE__); \
    }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUerror: %s\nCode: %d\nFile: %s\nLine: %d\n", hipGetErrorString(code), code, file, line);
        if (abort)
            exit(code);
    }
}

/*
    Entire sort kernel:
        1. Radix sort
        2. Merge sort
*/
__global__ void sort_kernel(long int *data, unsigned long n, unsigned offset, const unsigned long n_threads)
{
    // extern __shared__ long int sdata[];
    const unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Variables useful to compute the portion of array for each thread
    unsigned long start = tid * offset;
    unsigned long end = start + offset - 1;
    unsigned old_offset = 0;
    unsigned prec_thread = 0;

    // Variables useful during the merging phase
    unsigned long temp_n_threads = n_threads; // Variable useful to compute the numbers of levels during the merging phase
    unsigned level_merge = 0, levels_merge = 0, offset_merge = 0, threads_to_merge = 0;
    unsigned left = 0, mid = 0, right = 0;

    // Compute new start, end and offset for the thread, computing the offset of precedent threads
    if (tid != 0)
    {
        // Compute old offset in a recursive way, in order to compute the start for the current thread
        if (tid - 1 == 0)
        {
            start = tid * offset;
        }
        else
        {
            old_offset = offset;
            for (prec_thread = 1; prec_thread < tid; prec_thread++)
            {
                /*
                    This if-else is useful if there are more thread than needed:
                        - Ensures that no necessary thread remain in idle
                */
                if ((n - old_offset) > 0)
                {
                    // ceil((n - old_offset/n_threads - prec_thread))
                    old_offset += (n - old_offset + (n_threads - prec_thread) - 1) / (n_threads - prec_thread);
                }
                else
                {
                    break;
                }
            }
            start = old_offset;
        }

        // ceil((n - start) / (n_threads - tid))
        offset = (n - start + (n_threads - tid) - 1) / (n_threads - tid);
        end = start + offset - 1;
    }

    /*
        This if-else is useful if there are more thread than needed:
            - It ensures that no necessary thread remain in idle
    */
    if ((n - old_offset) > 0)
    {

        /*
            Log(num_threads)/Log(2) == Log_2(num_threads)
            Compute number of merge needed in the merge sort
        */
        while (temp_n_threads > 1)
        {
            temp_n_threads /= 2;
            levels_merge++;
        }

        // // Load data into shared memory
        // for (long i = start; i < end + 1; i++)
        // {
        //     sdata[i] = data[i];
        // }

        radix_sort(&data[start], offset);
        __syncthreads();

        // Merge - Phase
        for (level_merge = 1; level_merge <= levels_merge; level_merge++)
        {
            /*
                At first level, mid is equal to the end of the portion sorted by the thread since during the merging phase,
                mid is the final index of the left portion.
            */
            if (level_merge == 1)
            {
                mid = end;
            }

            /*
                threads_to_merge = 2^(level_merge) - Useful to exclude no necessary thread in the successive level
                Threads_to_merge is equal to the number of threads merged from the first level of the merging phase
            */
            power(2, level_merge, &threads_to_merge);

            if ((tid % threads_to_merge) == 0)
            {
                left = start;
                offset_merge = offset;

                /*
                    Useful to compute the size of the resulting list after the current level_merge
                */
                for (prec_thread = tid + 1; prec_thread < tid + threads_to_merge; prec_thread++)
                {
                    // ceil((n - start - offset_merge) / (n_threads - prec_thread))
                    offset_merge += (n - start - offset_merge + (n_threads - prec_thread) - 1) / (n_threads - prec_thread);
                }

                right = left + offset_merge - 1;
                merge(data, left, mid, right);

                /*
                    Merge sort expects to have mid as maximum value of the first list
                    Mid is equal to right to the next level_merge

                */
                mid = right;
            }

            // Needed since the lists to merge to the next level_merge must be ordered
            __syncthreads();
        }

        // // Write sorted data back to global memory
        // for (long i = start; i < start + offset && i < n; i++)
        // {
        //     data[i] = sdata[i];
        // }
    }
}

/*
    Function that returns the number of lists to merge at level 0 of the merging phase
*/
unsigned long get_n_list_to_merge(unsigned long n, unsigned partition, unsigned num_threads)
{
    unsigned thread = 0;
    unsigned long offset = partition, n_list_to_merge = 1;

    for (thread = 1; thread < num_threads; thread++)
    {
        if ((n - offset) > 0)
        {
            // ceil((n - offset) / (num_threads - ))
            offset += (n - offset + (num_threads - thread) - 1) / (num_threads - thread);
            n_list_to_merge++;
        }
        else
        {
            break;
        }
    }

    return n_list_to_merge;
}

// TODO:Comment this function
void get_start_and_size(unsigned long *block_dimension, unsigned long *offsets, unsigned long n, unsigned partition, unsigned total_blocks, unsigned total_threads)
{
    unsigned int idx_start = 0;
    unsigned int idx_size = 0;
    unsigned int idx_tid = 0; // Actual thread in the block

    unsigned long thread = 0;
    unsigned num_blocks_sort = total_threads / (float)MAXTHREADSPERBLOCK;
    unsigned multiplier = num_blocks_sort / (float)total_blocks;
    unsigned long precedent_threads = multiplier * MAXTHREADSPERBLOCK;
    unsigned current_block = 0;

    // unsigned long total_threads = (num_block + 1) * MAXTHREADSPERBLOCK;
    unsigned start_v = 0;
    unsigned size_v = 0;
    unsigned long offset = 0;

    // Initialization of the offset for each thread in each block
    for (unsigned i = 0; i < total_blocks * MAXTHREADSPERBLOCK; i++)
    {
        offsets[i] = 0;
    }

    for (current_block = 0; current_block < total_blocks; current_block++)
    {
        precedent_threads *= current_block;
        idx_start = current_block * 2;
        idx_size = idx_start + 1;
        idx_tid = current_block * MAXTHREADSPERBLOCK;

        if (current_block == 0)
        {
            start_v = 0;
        }
        else
        {
            // Compute start in a recursive way
            for (thread = 0; thread < precedent_threads; thread++)
            {
                if ((n - size_v) > 0) // MORE THREAD THAN NEEDED
                {
                    size_v = (n - start_v + (total_threads - thread) - 1) / (total_threads - thread);
                    start_v += size_v;
                }
                else
                {
                    break;
                }
            }
        }

        block_dimension[idx_start] = start_v;

        size_v = start_v;
        // Compute size in a recursive way
        for (thread = precedent_threads; thread < (current_block + 1) * MAXTHREADSPERBLOCK * multiplier; thread++)
        {
            if ((n - size_v) > 0) // MORE THREAD THAN NEEDED
            {
                offset = (n - size_v + (total_threads - thread) - 1) / (total_threads - thread);
                size_v += offset;
                offsets[idx_tid] += offset;
                if (((thread + 1) % multiplier) == 0)
                {
                    idx_tid++;
                }
            }
            else
            {
                break;
            }
        }

        block_dimension[idx_size] = size_v - start_v;
    }
}

// TODO:Comment this function
__global__ void merge_kernel(long int *data, unsigned long n, unsigned long *offset, const unsigned long n_threads, const unsigned current_block)
{
    // extern __shared__ long int sdata[];
    const unsigned tid = current_block + threadIdx.x;
    unsigned long start = 0;
    unsigned long end = 0;

    unsigned long left, mid, right, offset_merge;
    unsigned level_merge = 0, levels_merge = 0;
    unsigned temp_n_threads = n_threads;
    unsigned num_thread_to_merge = 0, threads_to_merge = 0;

    unsigned long i;

    // printf("OFFSET: %lu\n", offset[current_block]);

    // Compute new start, end and offset for the thread, computing the offset of precedent threads
    for (i = current_block; i < tid; i++)
    {
        start += offset[i];
    }

    // printf("TID: %lu - START: %li", tid, start);
    end = start + offset[tid] - 1;

    // Log(n_threads)/Log(2) == Log_2(n_threads)
    // Compute number of merge needed in the merge sort
    while (temp_n_threads > 1)
    {
        temp_n_threads /= 2;
        levels_merge++;
    }

    // printf("Sono il thread n.ro %lu con last n.ro %lu\n", start, end);

    // // Load data into shared memory
    // for (i = start; i < end + 1; i++)
    // {
    //     sdata[i] = data[i];
    // }

    // Merge the sorted array
    for (level_merge = 0; level_merge < levels_merge; level_merge++)
    {
        if (level_merge == 0)
        {
            mid = end;
        }

        power(2, level_merge, &threads_to_merge);

        if ((tid % threads_to_merge) == 0)
        {
            left = start;
            offset_merge = offset[tid]; //TODO: PROBLEM WITH OFFSET - AT STEP 2 IT BECAMES 0 - NO SENSE
            printf("STEP: %d - TID: %d - OFFSET_TID: %lu\n", level_merge, tid, offset[tid]);

            for (num_thread_to_merge = current_block + 1; num_thread_to_merge < current_block + threads_to_merge; num_thread_to_merge++)
            {
                offset_merge += offset[tid + num_thread_to_merge];
            }

            right = left + offset_merge - 1;
            // printf("STEP: %d - TID: %d - RIGHT: %d\n", level_merge, tid, right);
            // printf("STEP: %d - TID: %d - LEFT: %d\n", level_merge, tid, left);
            // printf("STEP: %d - TID: %d - OFFSET_MERGE: %d\n", level_merge, tid, offset_merge);
            // printf("MID: TID: %d-%d\n", tid, mid);
            merge(data, left, mid, right);
            // for (long k = start; k < left + offset_merge; k++)
            // {
            //     printf("%lu:%li\n", k, sdata[k]);
            // }

            // Fix since the two merged list are of two different dimension, because the offset is balanced between threads.
            // Merge sort expects to have mid as maximum value of the first list
            mid = right;
        }
        __syncthreads();
    }
}

int main(int argc, char *argv[])
{
    unsigned long N = 512;
    unsigned long first, last;
    long int *a, *dev_a;
    unsigned long n_threads_per_block = 0, n_blocks = 0, n_total_threads = 0;
    unsigned partition_size = 50; // TODO: TEMPORARY VALUE - TO CHECK OTHER VALUES
    double tstart = 0, tstop = 0;

    // Variables useful during the parallel sorting
    unsigned long n_merge = 0;
    unsigned n_blocks_merge = 0;

    // Variables useful to manage the partition of array to assign to each block during the merging phase
    unsigned idx_block_start = 0;
    unsigned idx_block_size = 0;
    unsigned long *block_dimension;

    // Variables useful to manage the partition of array to assign at each thread in each block at level 0 during the merging phase
    unsigned long *thread_offset, *dev_thread_offset, *block_offset;

    if (argc > 1)
    {
        N = atoi(argv[1]);
    }

    first = 0;
    last = N - 1;
    const size_t size_array = N * sizeof(long int);

    a = (long int *)malloc(size_array);
    cudaHandleError(hipMalloc((void **)&dev_a, size_array));

    // Sequential sorting
    printf("Sort algorithm on array of %lu elements\n\n", N);
    printf("Sequential implementation:\n");
    init_array(a, N);
    tstart = gettime();
    merge_sort(a, first, last);
    tstop = gettime();
    check_result(a, N);
    bzero(a, size_array); // Erase destination buffer
    printf("Elapsed time in seconds: %f\n\n", (tstop - tstart));

    // Parallel sorting
    printf("Parallel implementation:\n");
    init_array(a, N);
    cudaHandleError(hipMemcpy(dev_a, a, size_array, hipMemcpyHostToDevice));

    /*
        Ensures the minimum numbers of necessary thread
            - First branch: N is smaller or equal than the starting partition size of each thread
                - Starting from the maximum number of thread needed (N), it checks that the number of threads is a power of two,
                    otherwise the merging phase will not work
            - Second branch: N is greater than the starting partition size of each thread
                - It checks that the number of necessary threads is smaller or equal than the number of threads for each block
                    and it computes the partition size
                - If the number of necessary threads is smaller than the number of threads for each blocks,
                    it does the same thing of the first branch starting from the number of necessary thread
                - Otherwise it computes the number of minimum blocks needed ensuring that the number of threads is a power of 2

    */
    if (N <= partition_size)
    {
        n_blocks = 1; // TODO:Depends if the partition size is greater than MAXTHREADPERBLOCK
        for (unsigned long i = N; i >= 2; i--)
        {
            if (IsPowerOfTwo(i))
            {
                n_total_threads = i;
                partition_size = ceil(N / float(n_total_threads));
                n_threads_per_block = n_total_threads;
                break;
            }
        }
    }
    else
    {
        n_total_threads = ceil(N / float(partition_size));

        if (n_total_threads <= MAXTHREADSPERBLOCK)
        {
            n_blocks = 1;
            n_threads_per_block = n_total_threads;

            for (unsigned long i = n_total_threads; i >= 2; i--)
            {
                if (IsPowerOfTwo(i))
                {
                    n_total_threads = i;
                    partition_size = ceil(N / float(n_total_threads));
                    n_threads_per_block = n_total_threads;
                    break;
                }
            }
        }
        else
        {
            n_threads_per_block = MAXTHREADSPERBLOCK;
            n_blocks = ceil(n_total_threads / (float)n_threads_per_block);

            if (n_blocks > MAXBLOCKS)
            {
                n_blocks = MAXBLOCKS;
            }

            n_total_threads = (unsigned long)(n_blocks * n_threads_per_block);

            for (unsigned long i = n_total_threads; i >= 2; i--)
            {
                n_blocks = ceil(i / (float)MAXTHREADSPERBLOCK);
                n_total_threads = (unsigned long)(n_blocks * n_threads_per_block);

                if (IsPowerOfTwo(n_total_threads))
                {
                    partition_size = ceil(N / (float)n_total_threads);
                    break;
                }
            }
        }
    }

    dim3 blockSize(n_threads_per_block);
    dim3 gridSize(n_blocks);

    printf("NUM_THREADS: %lu\n", n_total_threads);
    printf("NUM BLOCKS: %lu\n", n_blocks);
    printf("NUM THREAD PER BLOCK: %lu\n", n_threads_per_block);

    n_merge = ceil(get_n_list_to_merge(N, partition_size, n_total_threads) / (float)2);
    n_blocks_merge = ceil(n_merge / (float)MAXTHREADSPERBLOCK);

    const size_t size_blocks = n_blocks_merge * MAXTHREADSPERBLOCK * sizeof(unsigned long);

    block_dimension = (unsigned long *)malloc(n_blocks_merge * 2 * sizeof(unsigned long));
    thread_offset = (unsigned long *)malloc(size_blocks);
    block_offset = (unsigned long *)malloc(n_blocks_merge * sizeof(unsigned long));
    cudaHandleError(hipMalloc((void **)&dev_thread_offset, size_blocks));

    // for (int i = 0; i < n_blocks_merge; i++)
    // {
    //     block_dimension[i] = (unsigned long *)malloc(2 * sizeof(unsigned long)); // 2 since the first position is the start and the second position is the size
    //     thread_offset[i] = (unsigned long *)malloc(size_threads);
    //     cudaHandleError(hipMallocManaged((void **)&dev_thread_offset[i], size_threads));
    // }

    tstart = gettime();
    // sort_kernel<<<gridSize, blockSize, size>>>(dev_a, N, partition_size, num_total_threads); //problem with size shared memory

    /*
        Two different branch to compute the parallel sorting based on the number of blocks
            - First branch: compute the radix sort phase and the merging sort phase in the same kernel
            - Second branch: compute the two phase in two distinct moments
                - The radix sort is computed on the entire array with the all necessary blocks
                - The sorting phase is computed using a different number of blocks, since the number of necessary threads is smaller
                    - By doing so all the threads in each block performs a merge during the first level of the merging phase
                    - Then, the sorting is called on only one block in order to sort all the portion of array sorted by each block
    */
    if (n_blocks == 1)
    {
        sort_kernel<<<gridSize, blockSize>>>(dev_a, N, partition_size, n_total_threads); // GLOBAL MEMORY
    }
    else // TODO: PROBLEM WITH 25601 - TEST ALSO WITH 100000
    {
        /*
        STEPS:
        0. Call the radix sort on the array - DONE
        1. Compute the numbers of list to merge - DONE
        2. Get a different portion of the array for each block - DONE
        3. Write a for-loop in which you call each block on a different portion of the array
        4. hipDeviceSynchronize();
        5. Call a single block to merge the entire array on the different results of the different blocks
        */

        // The data has to be ordered before merging phase
        radix_sort_kernel<<<gridSize, blockSize>>>(dev_a, N, partition_size, n_total_threads); // GLOBAL MEMORY; TODO: here I could use shared memory with size equal to partition_size
        cudaHandleError(hipDeviceSynchronize());
        cudaHandleError(hipPeekAtLastError());

        // Compute the size of dev_a and where to start
        get_start_and_size(block_dimension, thread_offset, N, partition_size, n_blocks_merge, n_total_threads);
        cudaHandleError(hipMemcpy(dev_thread_offset, thread_offset, size_blocks, hipMemcpyHostToDevice));

        printf("N BLOCKs MERGE: %d\n", n_blocks_merge);

        for (unsigned num_block = 0; num_block < n_blocks_merge; num_block++) // TODO: TEST WITH N=25601
        {
            idx_block_start = num_block * 2;
            idx_block_start = idx_block_start + 1;
            printf("NUM BLOCK MERGE: %d\n", num_block);

            // TODO: SICURO SI PUò USARE SHARED MEMORY SUGLI OFFSET
            merge_kernel<<<1, blockSize>>>(&dev_a[block_dimension[idx_block_start]], block_dimension[idx_block_size], dev_thread_offset, n_threads_per_block, num_block * MAXTHREADSPERBLOCK); // GLOBAL MEMORY;
            // cudaHandleError(hipPeekAtLastError());

            block_offset[num_block] = block_dimension[idx_block_size] - block_dimension[idx_block_start];
        }

        cudaHandleError(hipPeekAtLastError());
        cudaHandleError(hipDeviceSynchronize());

        merge_kernel<<<1, blockSize>>>(dev_a, N, block_offset, n_threads_per_block, 0); // GLOBAL MEMORY;
    }

    tstop = gettime();
    cudaHandleError(hipPeekAtLastError());
    cudaHandleError(hipMemcpy(a, dev_a, size_array, hipMemcpyDeviceToHost));
    check_result(a, N);
    bzero(a, size_array); // Erase destination buffer
    printf("Elapsed time in seconds: %f\n\n", (tstop - tstart));

    // Free memory on host and device
    free(a);
    free(block_dimension);
    free(thread_offset);
    free(block_offset);
    cudaHandleError(hipFree(dev_thread_offset));
    cudaHandleError(hipFree(dev_a));

    return 0;
}