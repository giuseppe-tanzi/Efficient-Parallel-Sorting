#include "hip/hip_runtime.h"
#include "../lib/parallelSort.cuh"

void parallel_sort(unsigned short *dev_a,
                   const unsigned long long N,
                   ParallelSortConfig config,
                   const size_t size_blocks,
                   const unsigned blocks_involved_in_merging,
                   unsigned long long *block_starting_idx,
                   unsigned long long *block_size,
                   unsigned long *thread_offset,
                   unsigned long *dev_thread_offset)
{

    if (config.total_blocks == 1)
    {
        /*
            - Compute the radix sort phase and the merging sort phase in the same kernel
        */
        sort_kernel<<<config.gridSize, config.blockSize>>>(dev_a, N, config.partition_size, config.total_threads); // GLOBAL MEMORY
    }
    else
    {
        /*
            - Compute the sorting in two different phase
        */

        /*
            - The radix sort is computed on the entire array with the all necessary blocks
        */
        radix_sort_kernel<<<config.gridSize, config.blockSize>>>(dev_a, N, config.partition_size, config.total_threads); // GLOBAL MEMORY; TODO: here I could use shared memory with size equal to partition_size
        cudaHandleError(hipDeviceSynchronize());
        cudaHandleError(hipPeekAtLastError());

        /*
            - Compute the start index on the data array for each block
            - Compute the offset on the data array to handle for each thread of each needed block
            - Compute the size of the data array to handle for each block
        */
        get_start_index_block(block_starting_idx, N, blocks_involved_in_merging, config.threads_per_block, config.total_threads);
        get_thread_offsets(thread_offset, block_starting_idx, N, blocks_involved_in_merging, config.threads_per_block, config.total_threads);
        get_size_block(block_size, block_starting_idx, N, blocks_involved_in_merging, config.threads_per_block, config.total_threads);

        cudaHandleError(hipMemcpy(dev_thread_offset, thread_offset, size_blocks, hipMemcpyHostToDevice));

        /*
            - The merging phase is computed using a different number of blocks, since the number of necessary threads is smaller
            - By doing so all the threads in each block performs a merge during the first level of the merging phase
            - Then, the sorting is called on only one block in order to sort all the portion of array sorted by each block
        */
        /*
            - It calls the merge kernel on each block
            - Each block has a defined portion of the array to handle and a precise number of lists to merge
            - The array will have blocks_involved_in_merging lists to merge at the end of the for-loop
        */
        for (unsigned block = 0; block < blocks_involved_in_merging; block++)
        {

            // TODO: SICURO SI PUò USARE SHARED MEMORY SUGLI OFFSET
            merge_kernel<<<1, config.blockSize>>>(dev_a + block_starting_idx[block], dev_thread_offset, config.threads_per_block, block * config.threads_per_block); // GLOBAL MEMORY;
        }
        
        /*
            - It performs the last merging phase with only one block since the number of lists to sort are surely
              less than the maximum thread number for each block
        */
        if (blocks_involved_in_merging > 1)
        {
            merge_blocks_kernel<<<1, blocks_involved_in_merging / 2>>>(dev_a, N, config, blocks_involved_in_merging / 2); // GLOBAL MEMORY;
        }
    }
}

__global__ void sort_kernel(unsigned short *data, const unsigned long long N, unsigned long long offset, const unsigned long total_threads)
{
    const unsigned long tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Variables useful to compute the portion of array for each thread
    unsigned long long start = tid * offset;
    unsigned long long end = start + offset - 1;
    unsigned long long old_offset = 0;
    unsigned long precedent_thread = 0;

    // Variables useful during the merging phase*
    unsigned long temp_total_threads = total_threads; // Variable useful to compute the numbers of levels during the merging phase
    unsigned level_merge = 0, levels_merge = 0, threads_to_merge = 0;
    unsigned long long offset_merge = 0;
    unsigned long long left = 0, mid = 0, right = 0;

    // Compute new start, end and offset for the thread, computing the offset of precedent threads
    if (tid != 0)
    {
        // Compute old offset in a recursive way, in order to compute the start for the current thread
        if (tid - 1 == 0)
        {
            start = tid * offset;
        }
        else
        {
            old_offset = offset;
            for (precedent_thread = 1; precedent_thread < tid; precedent_thread++)
            {
                /*
                    This if-else is useful if there are more thread than needed:
                        - Ensures that no necessary thread remain in idle
                */
                if ((N - old_offset) > 0)
                {
                    /*
                        - Compute old offset by computing how much offset the precedent threads have
                        - ceil((N - old_offset) / (total_threads - precedent_thread))
                    */  
                    old_offset += (N - old_offset + (total_threads - precedent_thread) - 1) / (total_threads - precedent_thread);
                }
                else
                {
                    break;
                }
            }
            start = old_offset;
        }

        /*
            - ceil((N - start) / (total_threads - tid))
        */
        offset = (N - start + (total_threads - tid) - 1) / (total_threads - tid);
        end = start + offset - 1;
    }

    /*
        This if-else is useful if there are more thread than needed:
            - It ensures that no necessary thread remain in idle
    */
    if ((N - old_offset) > 0)
    {

        /*
            Log(num_threads)/Log(2) == Log_2(num_threads)
            Compute number of merge needed in the merge sort
        */
        while (temp_total_threads > 1)
        {
            temp_total_threads /= 2;
            levels_merge++;
        }

        radix_sort_gpu(&data[start], offset);
        __syncthreads();

        // Merge - Phase
        for (level_merge = 1; level_merge <= levels_merge; level_merge++)
        {
            /*
                - At first level, mid is equal to the end of the portion sorted by the thread since during the merging phase,
                  mid is the final index of the left portion.
            */
            if (level_merge == 1)
            {
                mid = end;
            }

            /*
                - Threads_to_merge = 2^(level_merge) - Useful to exclude no necessary thread in the successive level
                - Threads_to_merge is equal to the number of threads merged from the first level of the merging phase
            */
            power(2, level_merge, &threads_to_merge);

            if ((tid % threads_to_merge) == 0)
            {
                left = start;
                offset_merge = offset;

                /*
                    Useful to compute the size of the resulting list after the current level_merge
                */
                for (precedent_thread = tid + 1; precedent_thread < tid + threads_to_merge; precedent_thread++)
                {
                    /*
                        - Compute offset_merge by computing how much offset the precedent threads have
                        - ceil((N - start - offset_merge) / (total_threads - precedent_thread))
                    */  
                    offset_merge += (N - start - offset_merge + (total_threads - precedent_thread) - 1) / (total_threads - precedent_thread);
                }

                right = left + offset_merge - 1;

                merge_gpu(data, left, mid, right);

                /*
                    - Merge sort expects to have mid as maximum value of the first list
                    - Mid is equal to right to the next level_merge
                */
                mid = right;
            }

            // Needed since the lists to merge to the next level_merge must be ordered
            __syncthreads();
        }
    }
}

__global__ void merge_kernel(unsigned short *data, const unsigned long *offset, const unsigned long total_threads, const unsigned long total_threads_precedent_blocks)
{
    const unsigned long tid = total_threads_precedent_blocks + threadIdx.x;
    unsigned long long start = 0;

    unsigned long long left, mid, right, offset_merge;
    unsigned level_merge = 0, levels_merge = 0;
    unsigned long temp_total_threads = total_threads;
    unsigned thread_to_merge = 0, threads_to_merge = 0;

    unsigned long i;

    // Compute the start for the thread, computing the offset of precedent threads
    for (i = total_threads_precedent_blocks; i < tid; i++)
    {
        start += offset[i];
    }

    /*
        - Log(total_threads)/Log(2) == Log_2(total_threads)
        - Compute number of merge needed in the merge sort
    */
    while (temp_total_threads > 1)
    {
        temp_total_threads /= 2;
        levels_merge++;
    }

    /*
        - Merge the sorted array
    */
    for (level_merge = 0; level_merge <= levels_merge; level_merge++)
    {
        power(2, level_merge, &threads_to_merge);

        if ((tid % threads_to_merge) == 0)
        {
            left = start;
            offset_merge = offset[tid];

            for (thread_to_merge = 1; thread_to_merge < threads_to_merge; thread_to_merge++)
            {
                offset_merge += offset[tid + thread_to_merge];
            }

            right = left + offset_merge - 1;

            if (level_merge == 0)
            {
                mid = left + (right - left) / 2;
            }

            merge_gpu(data, left, mid, right);

            /* 
                - Fix since the two merged list are of two different dimension, because the offset is balanced between threads
                - Merge sort expects to have mid as maximum value of the first list
            */
            mid = right;
        }
        __syncthreads();
    }
}

__global__ void merge_blocks_kernel(unsigned short *data, unsigned long long N, ParallelSortConfig config, const unsigned total_threads)
{
    const unsigned tid = threadIdx.x;
    unsigned long long start = 0;

    unsigned long long *block_starting_idx;
    unsigned long long *block_size;
    unsigned long long *thread_offset;
    unsigned long long *thread_mid;
    unsigned total_blocks = total_threads * 2;

    unsigned long long left, mid, right, offset_merge;
    unsigned level_merge = 0, levels_merge = 0;
    unsigned long temp_total_threads = total_threads;
    unsigned thread_to_merge = 0, total_threads_to_merge = 0;

    unsigned i;

    cudaHandleErrorGPU(hipMalloc((void **)&block_starting_idx, total_blocks * sizeof(unsigned long long)));
    cudaHandleErrorGPU(hipMalloc((void **)&block_size, total_blocks * sizeof(unsigned long long)));
    cudaHandleErrorGPU(hipMalloc((void **)&thread_offset, total_threads * sizeof(unsigned long long)));
    cudaHandleErrorGPU(hipMalloc((void **)&thread_mid, total_threads * sizeof(unsigned long long)));

    /*
        - Compute the start index on the data array for each block
        - Compute the size of the data array to handle for each block
    */
    get_start_index_block(block_starting_idx, N, total_blocks, config.threads_per_block, config.total_threads);
    get_size_block(block_size, block_starting_idx, N, total_blocks, config.threads_per_block, config.total_threads);

    for (unsigned block = 0; block < total_blocks; block++)
    {

        if ((block % 2) == 0)
        {
            // Add the offset of the successive block
            thread_offset[block / 2] = block_size[block] + block_size[block + 1];

            // Compute mid useful during the first level merge
            thread_mid[block / 2] = 0;
            for (i = 0; i <= block; i++)
            {
                thread_mid[block / 2] += block_size[i];
            }
        }
    }

    // Compute new start, end and offset for the thread, computing the offset of precedent threads
    for (unsigned thread = 0; thread < tid; thread++)
    {
        start += thread_offset[thread];
    }

    mid = thread_mid[tid] - 1;

    /*
        - Log(n_threads)/Log(2) == Log_2(n_threads)
        - Compute number of merge needed in the merge sort
    */
    while (temp_total_threads > 1)
    {
        temp_total_threads /= 2;
        levels_merge++;
    }

    /*
        - Merge the sorted array
    */
    for (level_merge = 0; level_merge <= levels_merge; level_merge++)
    {

        power(2, level_merge, &total_threads_to_merge);

        if ((tid % total_threads_to_merge) == 0)
        {
            left = start;
            offset_merge = thread_offset[tid];

            for (thread_to_merge = 1; thread_to_merge < total_threads_to_merge; thread_to_merge++)
            {
                offset_merge += thread_offset[tid + thread_to_merge];
            }

            right = left + offset_merge - 1;

            merge_gpu(data, left, mid, right);

            /*
                - Fix since the two merged list are of two different dimension, because the offset is balanced between threads.
                - Merge sort expects to have mid as maximum value of the first list
            */
            mid = right;
        }
        __syncthreads();
    }

    cudaHandleErrorGPU(hipFree(block_starting_idx));
    cudaHandleErrorGPU(hipFree(block_size));
    cudaHandleErrorGPU(hipFree(thread_offset));
    cudaHandleErrorGPU(hipFree(thread_mid));
}