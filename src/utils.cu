#include "hip/hip_runtime.h"
#include "../lib/utils.cuh"

void gpuAssert(hipError_t code, const char *file, int line, bool abort)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUerror: %s\nCode: %d\nFile: %s\nLine: %d\n", hipGetErrorString(code), code, file, line);
        if (abort)
            exit(code);
    }
}

double gettime(void)
{
    struct timespec ts;
    clock_gettime(CLOCK_MONOTONIC, &ts);
    return (ts.tv_sec + (double)ts.tv_nsec / 1e9);
}

void init_array(unsigned short *data, const unsigned long long N)
{
    srand(42); // Ensure the determinism

    for (unsigned long long i = 0; i < N; i++)
    {
        data[i] = rand() % (MAX_VALUE - MIN_VALUE + 1) + MIN_VALUE;;
    }
}

__host__ __device__ void print_array(const unsigned short *data, const unsigned long long N) //TODO: delete __device__
{
    for (unsigned long long i = 0; i < N; i++)
    {
        printf("%hu ", data[i]);
    }
    printf("\n");
}

int check_result(unsigned short *results, const unsigned long long N)
{
    for (unsigned long long i = 0; i < N - 1; i++)
    {
        if (results[i] > results[i + 1])
        {
            printf("Check failed: data[%llu] = %hu, data[%llu] = %hu\n", i, results[i], i + 1, results[i + 1]);
            printf("%hu is greater than %hu\n", results[i], results[i + 1]);
            return 0;
        }
    }
    printf("Check OK\n");
    return 1;
}

bool IsPowerOfTwo(const unsigned long x)
{
    return (x & (x - 1)) == 0;
}

__device__ void get_max(unsigned short *data, const unsigned long long N, unsigned short *max)
{
    *max = -INFINITY;
    for (unsigned long long i = 0; i < N; i++)
    {
        if (data[i] > *max)
        {
            *max = data[i];
        }
    }
}

__device__ void power(unsigned base, unsigned exp, unsigned *result)
{
    *result = 1;
    for (;;)
    {
        if (exp & 1)
            *result *= base;
        exp >>= 1;
        if (!exp)
            break;
        base *= base;
    }
}