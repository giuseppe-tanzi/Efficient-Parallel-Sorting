#include "hip/hip_runtime.h"
#include "../lib/utils.cuh"

void gpuAssert(hipError_t code, const char *file, int line, bool abort)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUerror: %s\nCode: %d\nFile: %s\nLine: %d\n", hipGetErrorString(code), code, file, line);
        if (abort)
            exit(code);
    }
}

__device__ void gpuAssert_dev(hipError_t code, const char *file, int line, bool abort)
{
    if (code != hipSuccess)
    {
        const char* errorString = hipGetErrorString(code);

        printf("GPUerror: %s\nCode: %d\nFile: %s\nLine: %d\n", errorString, code, file, line);

        if (abort)
            asm("trap;");
    }
}


double get_time(void)
{
    struct timespec ts;
    clock_gettime(CLOCK_MONOTONIC, &ts);
    return (ts.tv_sec + (double)ts.tv_nsec / 1e9);
}

void init_array(unsigned short *data, const unsigned long long N)
{
    srand(42); // Ensure the determinism

    for (unsigned long long i = 0; i < N; i++)
    {
        data[i] = rand() % (MAX_VALUE - MIN_VALUE + 1) + MIN_VALUE;;
    }
}

__host__ void print_array(const unsigned short *data, const unsigned long long N)
{
    for (unsigned long long i = 0; i < N; i++)
    {
        printf("%hu ", data[i]);
    }
    printf("\n");
}

int check_result(unsigned short *result, const unsigned long long N)
{
    for (unsigned long long i = 0; i < N - 1; i++)
    {
        if (result[i] > result[i + 1])
        {
            printf("Check failed: data[%llu] = %hu, data[%llu] = %hu\n", i, result[i], i + 1, result[i + 1]);
            printf("%hu is greater than %hu\n", result[i], result[i + 1]);
            return 0;
        }
    }
    printf("Check OK\n");
    return 1;
}

bool is_power_of_two(const unsigned long x)
{
    return (x & (x - 1)) == 0;
}

__device__ void get_max(unsigned short *data, const unsigned long long N, unsigned short *max)
{
    *max = -INFINITY;
    for (unsigned long long i = 0; i < N; i++)
    {
        if (data[i] > *max)
        {
            *max = data[i];
        }
    }
}

__device__ void power(unsigned base, unsigned exp, unsigned *result)
{
    *result = 1;
    for (;;)
    {
        if (exp & 1)
            *result *= base;
        exp >>= 1;
        if (!exp)
            break;
        base *= base;
    }
}